#include "hip/hip_runtime.h"
#include <Grid.hpp>
#include <hipcub/hipcub.hpp>

__global__ void initGridKernel(const Particle*, uint*, uint*);
__global__ void updateGridKernel(const uint*, int*);
__global__ void sortParticleBuffer(uint* particle_ID, Particle* particles, Particle* temp);

Grid::Grid(float3 volume, float kernel_radius, uint num_parts) : num_particles(num_parts) {
    grid_size = {
        static_cast<uint>(ceil(volume.x / kernel_radius)),
        static_cast<uint>(ceil(volume.y / kernel_radius)),
        static_cast<uint>(ceil(volume.z / kernel_radius)),
    };
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(GRID_DIMS), &grid_size, sizeof(uint3)));

    // malloc grid main arrays
    CHECK(hipMalloc(&grid_cell_ID, num_particles * sizeof(uint)));
    CHECK(hipMalloc(&particle_ID, num_particles * sizeof(uint)));
    CHECK(hipMalloc(&cellStartIdx, getLinearGridSize() * sizeof(int)));
    CHECK(hipMalloc(&d_temp_particle, num_particles * sizeof(Particle)));

    // malloc support arrays needed for sorting the grid
    CHECK(hipMalloc(&d_keys_out, num_particles * sizeof(uint)));
    CHECK(hipMalloc(&d_values_out, num_particles * sizeof(uint)));

    CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, d_storage_size, grid_cell_ID, d_keys_out, particle_ID, d_values_out, num_particles));
    CHECK(hipMalloc(&d_temp_storage, d_storage_size));
    printf("LinearGridSize: %d\n", getLinearGridSize());
}

Grid::~Grid() {
    CHECK(hipFree(grid_cell_ID));
    CHECK(hipFree(particle_ID));
    CHECK(hipFree(cellStartIdx));
    CHECK(hipFree(d_temp_storage));
    CHECK(hipFree(d_keys_out));
    CHECK(hipFree(d_values_out));
    CHECK(hipFree(d_temp_particle));
}
void Grid::sortGrid() {
    CHECK(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, d_storage_size, grid_cell_ID, d_keys_out, particle_ID, d_values_out, num_particles));
    CHECK(hipMemcpy(grid_cell_ID, d_keys_out, num_particles * sizeof(uint), hipMemcpyDeviceToDevice));
    CHECK(hipMemcpy(particle_ID, d_values_out, num_particles * sizeof(uint), hipMemcpyDeviceToDevice));
}

void Grid::updateGrid(Particle* particles) {
    int blockSize = 256;
    int numBlocks = (num_particles + blockSize - 1) / blockSize;

    CHECK(hipMemset(cellStartIdx, -1, getLinearGridSize() * sizeof(int)));

    initGridKernel<<<numBlocks, blockSize>>>(particles, grid_cell_ID, particle_ID);
    CHECK(hipDeviceSynchronize());

    sortGrid();

    updateGridKernel<<<numBlocks, blockSize>>>(grid_cell_ID, cellStartIdx);
    CHECK(hipDeviceSynchronize());

    sortParticleBuffer<<<numBlocks, blockSize>>>(particle_ID, particles, d_temp_particle);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(particles, d_temp_particle, num_particles * sizeof(Particle), hipMemcpyDeviceToDevice));
}

__global__ void sortParticleBuffer(uint* particle_ID, Particle* particles, Particle* temp) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUM_PARTICLES) {
        temp[idx] = particles[particle_ID[idx]];
        particle_ID[idx] = idx;
    }
}

__global__ void initGridKernel(const Particle* particles, uint* grid_cell, uint* part_ID) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUM_PARTICLES) {
        grid_cell[idx] = linearGridIndex(getGridCoordinate(particles[idx]));
        part_ID[idx] = idx;
    }
}

__global__ void updateGridKernel(const uint* grid_cell, int* cellStart) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
        cellStart[0] = grid_cell[0];
    if (idx < NUM_PARTICLES - 1)
        if (grid_cell[idx] != grid_cell[idx + 1])
            cellStart[grid_cell[idx + 1]] = idx + 1;
}
